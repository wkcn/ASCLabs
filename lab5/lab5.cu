#include <iostream>
#include <fstream>
#include <ctime>
#include <hip/hip_runtime.h>

using namespace std;

const int N = 400;
const int P = 600;
const int M = 200;

float A[N][P];
float B[P][M];
float C[N][M];

float *A_D,*B_D,*C_D;

void Init_Data(){
	//Mat A
	for (int i = 0;i < N;++i){
		for (int j = 0;j < P;++j){
			A[i][j] = (j+1) * 1.f / (N * (i+1));
		}
	}
	//Mat B
	for (int i = 0;i < P ;++i){
		for (int j = 0;j  < M;++j){
			B[i][j] = M * 1.0 / ((i+1) * (j+1));
		}
	}
	//Copy data to cuda
	hipMalloc((void **)&A_D, sizeof(float) * N * P);
	hipMalloc((void **)&B_D, sizeof(float) * P * M);
	hipMalloc((void **)&C_D, sizeof(float) * N * M);
	hipMemcpy(A_D, (void*)A, sizeof(float) * N * P, hipMemcpyHostToDevice);
	hipMemcpy(B_D, (void*)B, sizeof(float) * P * M, hipMemcpyHostToDevice);
}

__global__ void MatMul(float *A_D, float *B_D, float *C_D){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if (i >= N || j >= M)return;
	
	float v = 0;
	for (int k = 0;k < P;++k){
		v += A_D[i * P + k] * B_D[k * M + j];
 	}
	C_D[i * M + j] = v;
	
} 

void Output(){
	hipMemcpy((void*)C, C_D, sizeof(float) * N * M, hipMemcpyDeviceToHost); 
	ofstream fout("cudaresult.txt");
	for (int i = 0;i < N;++i){
		for (int j = 0;j < M;++j){
			if (j != 0){
				fout << " ";
			}
			fout << C[i][j];
		}
		fout << endl;
	}
}

int main(){
	Init_Data();
	dim3 dimGrid(20, 20);
	//dim3 dimBlock(N,M);
	//dim3 dimBlock((N+31)/32*32,(M+31)/32*32);
	dim3 dimBlock(20,10);
	clock_t t = clock();
	MatMul<<<dimGrid, dimBlock>>>(A_D, B_D, C_D);
	cout << "Cuda Used Time: "<< double((clock() - t)*1.0/CLOCKS_PER_SEC) << endl;
	Output();

	//Release Source
	hipFree(A_D);
	hipFree(B_D);
	hipFree(C_D);
	return 0;
}
